// GPUWrapper.cu
#include <hip/hip_runtime.h>
#include <vector>
#include <string>
#include <cstring>
#include <stdexcept>
#include <iostream>
#include <iomanip>

#include "GPUSHA512.cuh"
#include "GPUPBKDF2.cuh"
#include "GPUHMAC.cuh"

#define BITCOIN_SEED "Bitcoin seed"
#define SHA512_DIGEST_SIZE 64
#define SEED_SIZE 64
#define HMAC_BLOCK_SIZE 128

// === CUDA 错误包装宏 ===
#define CudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
inline void __cudaSafeCall(hipError_t err, const char *file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

typedef std::vector<unsigned char> ByteVec;

__host__ std::pair<ByteVec, ByteVec> derive_master_key(const std::string& mnemonic, const std::string& passphrase) {
    initSHA512Constants();

    // 1. === 生成 seed (PBKDF2 with HMAC-SHA512) ===
    std::string salt = "mnemonic" + passphrase;
    char *d_mnemonic, *d_salt;
    BYTE *d_seed;
    CudaSafeCall(hipMalloc(&d_mnemonic, mnemonic.size()));
    CudaSafeCall(hipMalloc(&d_salt, salt.size()));
    CudaSafeCall(hipMalloc(&d_seed, SEED_SIZE));

    CudaSafeCall(hipMemcpy(d_mnemonic, mnemonic.data(), mnemonic.size(), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_salt, salt.data(), salt.size(), hipMemcpyHostToDevice));

    pbkdf2_kernel<<<1, 1>>>(d_mnemonic, mnemonic.size(), d_salt, salt.size(), PBKDF2_HMAC_SHA512_ITERATIONS, d_seed);
    CudaSafeCall(hipDeviceSynchronize());

    ByteVec seed(SEED_SIZE);
    CudaSafeCall(hipMemcpy(seed.data(), d_seed, SEED_SIZE, hipMemcpyDeviceToHost));

    CudaSafeCall(hipFree(d_mnemonic));
    CudaSafeCall(hipFree(d_salt));
    CudaSafeCall(hipFree(d_seed));

    // 2. === HMAC-SHA512(seed, key="Bitcoin seed") ===
    const char* hmac_key = BITCOIN_SEED;
    size_t key_len = strlen(hmac_key);

    BYTE *d_key, *d_data, *d_hmac_out;
    CudaSafeCall(hipMalloc(&d_key, key_len));
    CudaSafeCall(hipMalloc(&d_data, SEED_SIZE));
    CudaSafeCall(hipMalloc(&d_hmac_out, SHA512_DIGEST_SIZE));

    CudaSafeCall(hipMemcpy(d_key, hmac_key, key_len, hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_data, seed.data(), SEED_SIZE, hipMemcpyHostToDevice));

    hmac_sha512_kernel<<<1, 1>>>((const char*)d_key, key_len, d_data, SEED_SIZE, d_hmac_out);
    CudaSafeCall(hipDeviceSynchronize());

    ByteVec I(SHA512_DIGEST_SIZE);
    CudaSafeCall(hipMemcpy(I.data(), d_hmac_out, SHA512_DIGEST_SIZE, hipMemcpyDeviceToHost));

    CudaSafeCall(hipFree(d_key));
    CudaSafeCall(hipFree(d_data));
    CudaSafeCall(hipFree(d_hmac_out));

    if (I.size() != 64) throw std::runtime_error("HMAC result is not 64 bytes");

    return {
        ByteVec(I.begin(), I.begin() + 32), // master private key
        ByteVec(I.begin() + 32, I.end())    // chain code
    };
}

__host__ std::vector<unsigned char> hmac_sha512(
    const std::vector<unsigned char>& key,
    const std::vector<unsigned char>& data,
    bool prehash_key
) {
    BYTE *d_key = nullptr, *d_data = nullptr, *d_out = nullptr;
    size_t final_key_len = key.size();

    if (prehash_key) {
        BYTE *d_tmp_in = nullptr, *d_tmp_out = nullptr;
        CudaSafeCall(hipMalloc(&d_tmp_in, key.size()));
        CudaSafeCall(hipMalloc(&d_tmp_out, SHA512_DIGEST_SIZE));
        CudaSafeCall(hipMemcpy(d_tmp_in, key.data(), key.size(), hipMemcpyHostToDevice));

        sha512_kernel<<<1, 1>>>(d_tmp_in, key.size(), d_tmp_out);
        CudaSafeCall(hipDeviceSynchronize());

        CudaSafeCall(hipMalloc(&d_key, SHA512_DIGEST_SIZE));
        CudaSafeCall(hipMemcpy(d_key, d_tmp_out, SHA512_DIGEST_SIZE, hipMemcpyDeviceToDevice));
        final_key_len = SHA512_DIGEST_SIZE;

        CudaSafeCall(hipFree(d_tmp_in));
        CudaSafeCall(hipFree(d_tmp_out));
    } else {
        CudaSafeCall(hipMalloc(&d_key, key.size()));
        CudaSafeCall(hipMemcpy(d_key, key.data(), key.size(), hipMemcpyHostToDevice));
    }

    CudaSafeCall(hipMalloc(&d_data, data.size()));
    CudaSafeCall(hipMemcpy(d_data, data.data(), data.size(), hipMemcpyHostToDevice));
    CudaSafeCall(hipMalloc(&d_out, SHA512_DIGEST_SIZE));

    hmac_sha512_kernel<<<1, 1>>>((const char*)d_key, final_key_len, d_data, data.size(), d_out);
    CudaSafeCall(hipDeviceSynchronize());

    std::vector<unsigned char> result(SHA512_DIGEST_SIZE);
    CudaSafeCall(hipMemcpy(result.data(), d_out, SHA512_DIGEST_SIZE, hipMemcpyDeviceToHost));

    CudaSafeCall(hipFree(d_key));
    CudaSafeCall(hipFree(d_data));
    CudaSafeCall(hipFree(d_out));

    return result;
}


#undef BITCOIN_SEED
#undef SHA512_DIGEST_SIZE
#undef SEED_SIZE
#undef HMAC_BLOCK_SIZE
