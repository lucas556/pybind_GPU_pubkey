// GPUWrapper.cu

#include <hip/hip_runtime.h>
#include <vector>
#include <string>
#include <cstring>
#include <stdexcept>
#include <iostream>
#include <iomanip>

#include "GPUSHA512.cuh"
#include "GPUPBKDF2.cuh"
#include "GPUHMAC.cuh"

#define BITCOIN_SEED "Bitcoin seed"
#define SHA512_DIGEST_SIZE 64
#define SEED_SIZE 64
#define HMAC_BLOCK_SIZE 128

typedef std::vector<unsigned char> ByteVec;

__host__ std::pair<ByteVec, ByteVec> derive_master_key(const std::string& mnemonic, const std::string& passphrase) {
    initSHA512Constants();

    // 1. === 生成 seed (PBKDF2 with HMAC-SHA512) ===
    std::string salt = "mnemonic" + passphrase;
    char *d_mnemonic, *d_salt;
    BYTE *d_seed;
    hipMalloc(&d_mnemonic, mnemonic.size());
    hipMalloc(&d_salt, salt.size());
    hipMalloc(&d_seed, SEED_SIZE);

    hipMemcpy(d_mnemonic, mnemonic.data(), mnemonic.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_salt, salt.data(), salt.size(), hipMemcpyHostToDevice);

    pbkdf2_kernel<<<1, 1>>>(d_mnemonic, mnemonic.size(), d_salt, salt.size(), PBKDF2_HMAC_SHA512_ITERATIONS, d_seed);
    hipDeviceSynchronize();

    ByteVec seed(SEED_SIZE);
    hipMemcpy(seed.data(), d_seed, SEED_SIZE, hipMemcpyDeviceToHost);

    hipFree(d_mnemonic);
    hipFree(d_salt);
    hipFree(d_seed);

    // 2. === HMAC-SHA512(seed, key="Bitcoin seed") ===
    const char* hmac_key = BITCOIN_SEED;
    size_t key_len = strlen(hmac_key);

    BYTE *d_key, *d_data, *d_hmac_out;
    hipMalloc(&d_key, key_len);
    hipMalloc(&d_data, SEED_SIZE);
    hipMalloc(&d_hmac_out, SHA512_DIGEST_SIZE);

    hipMemcpy(d_key, hmac_key, key_len, hipMemcpyHostToDevice);
    hipMemcpy(d_data, seed.data(), SEED_SIZE, hipMemcpyHostToDevice);

    hmac_sha512_kernel<<<1, 1>>>((const char*)d_key, key_len, d_data, SEED_SIZE, d_hmac_out);
    hipDeviceSynchronize();

    ByteVec I(SHA512_DIGEST_SIZE);
    hipMemcpy(I.data(), d_hmac_out, SHA512_DIGEST_SIZE, hipMemcpyDeviceToHost);

    hipFree(d_key);
    hipFree(d_data);
    hipFree(d_hmac_out);

    if (I.size() != 64) throw std::runtime_error("HMAC result is not 64 bytes");

    return {
        ByteVec(I.begin(), I.begin() + 32), // master private key
        ByteVec(I.begin() + 32, I.end())    // chain code
    };
}

__host__ std::vector<unsigned char> hmac_sha512(
    const std::vector<unsigned char>& key,
    const std::vector<unsigned char>& data,
    bool prehash_key
) {
    BYTE *d_key = nullptr, *d_data = nullptr, *d_out = nullptr;
    size_t final_key_len = key.size();

    if (prehash_key) {
        BYTE *d_tmp_in = nullptr, *d_tmp_out = nullptr;
        hipMalloc(&d_tmp_in, key.size());
        hipMalloc(&d_tmp_out, SHA512_DIGEST_SIZE);
        hipMemcpy(d_tmp_in, key.data(), key.size(), hipMemcpyHostToDevice);

        sha512_kernel<<<1, 1>>>(d_tmp_in, key.size(), d_tmp_out);
        hipDeviceSynchronize();

        hipMalloc(&d_key, SHA512_DIGEST_SIZE);
        hipMemcpy(d_key, d_tmp_out, SHA512_DIGEST_SIZE, hipMemcpyDeviceToDevice);
        final_key_len = SHA512_DIGEST_SIZE;

        hipFree(d_tmp_in);
        hipFree(d_tmp_out);
    } else {
        hipMalloc(&d_key, key.size());
        hipMemcpy(d_key, key.data(), key.size(), hipMemcpyHostToDevice);
    }

    hipMalloc(&d_data, data.size());
    hipMemcpy(d_data, data.data(), data.size(), hipMemcpyHostToDevice);
    hipMalloc(&d_out, SHA512_DIGEST_SIZE);

    hmac_sha512_kernel<<<1, 1>>>((const char*)d_key, final_key_len, d_data, data.size(), d_out);
    hipDeviceSynchronize();

    std::vector<unsigned char> result(SHA512_DIGEST_SIZE);
    hipMemcpy(result.data(), d_out, SHA512_DIGEST_SIZE, hipMemcpyDeviceToHost);

    hipFree(d_key);
    hipFree(d_data);
    hipFree(d_out);

    return result;
}

