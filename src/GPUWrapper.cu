#include "hip/hip_runtime.h"
// GPUWrapper.cu
#include <hip/hip_runtime.h>
#include <vector>
#include <string>
#include <cstring>
#include <stdexcept>
#include <iostream>
#include <iomanip>

#include "GPUSHA512.cuh"
#include "GPUPBKDF2.cuh"
#include "GPUHMAC.cuh"

#define BITCOIN_SEED "Bitcoin seed"
#define SHA512_DIGEST_SIZE 64
#define SEED_SIZE 64
#define HMAC_BLOCK_SIZE 128

// === CUDA 错误包装宏 ===
#define CudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
inline void __cudaSafeCall(hipError_t err, const char *file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

typedef std::vector<unsigned char> ByteVec;

__host__ std::pair<ByteVec, ByteVec> derive_master_key(const std::string& mnemonic, const std::string& passphrase) {
    initSHA512Constants();

    // 1. === 生成 seed (PBKDF2 with HMAC-SHA512) ===
    std::string salt = "mnemonic" + passphrase;
    char *d_mnemonic, *d_salt;
    BYTE *d_seed;
    CudaSafeCall(hipMalloc(&d_mnemonic, mnemonic.size()));
    CudaSafeCall(hipMalloc(&d_salt, salt.size()));
    CudaSafeCall(hipMalloc(&d_seed, SEED_SIZE));

    CudaSafeCall(hipMemcpy(d_mnemonic, mnemonic.data(), mnemonic.size(), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_salt, salt.data(), salt.size(), hipMemcpyHostToDevice));

    pbkdf2_kernel<<<1, 1>>>(d_mnemonic, mnemonic.size(), d_salt, salt.size(), PBKDF2_HMAC_SHA512_ITERATIONS, d_seed);
    CudaSafeCall(hipDeviceSynchronize());

    ByteVec seed(SEED_SIZE);
    CudaSafeCall(hipMemcpy(seed.data(), d_seed, SEED_SIZE, hipMemcpyDeviceToHost));

    CudaSafeCall(hipFree(d_mnemonic));
    CudaSafeCall(hipFree(d_salt));
    CudaSafeCall(hipFree(d_seed));

    // 2. === HMAC-SHA512(seed, key="Bitcoin seed") ===
    const char* hmac_key = BITCOIN_SEED;
    size_t key_len = strlen(hmac_key);

    BYTE *d_key, *d_data, *d_hmac_out;
    CudaSafeCall(hipMalloc(&d_key, key_len));
    CudaSafeCall(hipMalloc(&d_data, SEED_SIZE));
    CudaSafeCall(hipMalloc(&d_hmac_out, SHA512_DIGEST_SIZE));

    CudaSafeCall(hipMemcpy(d_key, hmac_key, key_len, hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_data, seed.data(), SEED_SIZE, hipMemcpyHostToDevice));

    hmac_sha512_kernel<<<1, 1>>>((const char*)d_key, key_len, d_data, SEED_SIZE, d_hmac_out);
    CudaSafeCall(hipDeviceSynchronize());

    ByteVec I(SHA512_DIGEST_SIZE);
    CudaSafeCall(hipMemcpy(I.data(), d_hmac_out, SHA512_DIGEST_SIZE, hipMemcpyDeviceToHost));

    CudaSafeCall(hipFree(d_key));
    CudaSafeCall(hipFree(d_data));
    CudaSafeCall(hipFree(d_hmac_out));

    if (I.size() != 64) throw std::runtime_error("HMAC result is not 64 bytes");

    return {
        ByteVec(I.begin(), I.begin() + 32), // master private key
        ByteVec(I.begin() + 32, I.end())    // chain code
    };
}

__global__ void ckd_data_kernel(
    const uint8_t* left_or_pubkey, bool hardened,
    uint32_t index, uint8_t* data_out
) {
    int offset = 0;

    if (hardened) {
        data_out[offset++] = 0x00;
        for (int i = 0; i < 32; ++i)
            data_out[offset++] = left_or_pubkey[i];
    } else {
        for (int i = 0; i < 33; ++i)
            data_out[offset++] = left_or_pubkey[i];
    }

    data_out[offset++] = (index >> 24) & 0xFF;
    data_out[offset++] = (index >> 16) & 0xFF;
    data_out[offset++] = (index >> 8) & 0xFF;
    data_out[offset++] = index & 0xFF;
}

// host端封装
std::vector<unsigned char> hmac_sha512_data(
    const std::vector<unsigned char>& key,
    const std::vector<unsigned char>& left_or_pubkey,
    bool hardened,
    uint32_t index
) {
    BYTE *d_key, *d_data, *d_out, *d_left;
    int data_len = hardened ? (1 + 32 + 4) : (33 + 4);

    // 1. GPU 内存分配
    CudaSafeCall(hipMalloc(&d_key, key.size()));
    CudaSafeCall(hipMemcpy(d_key, key.data(), key.size(), hipMemcpyHostToDevice));
    CudaSafeCall(hipMalloc(&d_left, left_or_pubkey.size()));
    CudaSafeCall(hipMemcpy(d_left, left_or_pubkey.data(), left_or_pubkey.size(), hipMemcpyHostToDevice));
    CudaSafeCall(hipMalloc(&d_data, data_len));
    CudaSafeCall(hipMalloc(&d_out, SHA512_DIGEST_SIZE));

    // 2. 构造 CKD 数据
    ckd_data_kernel<<<1, 1>>>(d_left, hardened, index, d_data);
    CudaSafeCall(hipDeviceSynchronize());

    // 3. 执行 HMAC
    hmac_sha512_kernel<<<1, 1>>>((const char*)d_key, key.size(), d_data, data_len, d_out);
    CudaSafeCall(hipDeviceSynchronize());

    // 4. 读取结果
    std::vector<unsigned char> result(SHA512_DIGEST_SIZE);
    CudaSafeCall(hipMemcpy(result.data(), d_out, SHA512_DIGEST_SIZE, hipMemcpyDeviceToHost));

    // 5. 清理资源
    hipFree(d_key); hipFree(d_left); hipFree(d_data); hipFree(d_out);

    return result;
}

#undef BITCOIN_SEED
#undef SHA512_DIGEST_SIZE
#undef SEED_SIZE
#undef HMAC_BLOCK_SIZE
