#include <iostream>
#include <vector>
#include <string>
#include <cstring>
#include <stdexcept>
#include <secp256k1.h>
#include <iomanip>

#include "GPUSHA512.cuh"
#include "GPUPBKDF2.cuh"
#include "GPUHMAC.cuh"

#define BITCOIN_SEED "Bitcoin seed"

__host__ std::vector<unsigned char> derive_pubkey(const std::vector<unsigned char>& privkey, secp256k1_context* ctx, bool compressed = true) {
    if (!ctx || privkey.size() != 32 || !secp256k1_ec_seckey_verify(ctx, privkey.data()))
        throw std::runtime_error("Invalid private key");

    secp256k1_pubkey pubkey;
    if (!secp256k1_ec_pubkey_create(ctx, &pubkey, privkey.data()))
        throw std::runtime_error("Pubkey creation failed");

    std::vector<unsigned char> output(compressed ? 33 : 65);
    size_t len = output.size();
    if (!secp256k1_ec_pubkey_serialize(ctx, output.data(), &len, &pubkey,
        compressed ? SECP256K1_EC_COMPRESSED : SECP256K1_EC_UNCOMPRESSED))
        throw std::runtime_error("Pubkey serialization failed");

    return output;
}

__host__ std::vector<unsigned char> add_privkeys_mod_n(const std::vector<unsigned char>& a, const std::vector<unsigned char>& b, secp256k1_context* ctx) {
    std::vector<unsigned char> out = a;
    if (!secp256k1_ec_seckey_tweak_add(ctx, out.data(), b.data()))
        throw std::runtime_error("secp256k1_ec_seckey_tweak_add failed");
    return out;
}

__host__ void gpu_pbkdf2_batch(const std::vector<std::string>& mnemonics, const std::string& passphrase, std::vector<ByteVec>& out_seeds, int threads_per_block) {
    int count = mnemonics.size();
    std::vector<std::string> salts(count, "mnemonic" + passphrase);

    std::vector<const char*> h_mnemonics(count), h_salts(count);
    std::vector<size_t> mnemonic_lens(count), salt_lens(count);
    for (int i = 0; i < count; ++i) {
        h_mnemonics[i] = mnemonics[i].c_str();
        h_salts[i] = salts[i].c_str();
        mnemonic_lens[i] = mnemonics[i].size();
        salt_lens[i] = salts[i].size();
    }

    char **d_mnemonics, **d_salts;
    size_t *d_mnemonic_lens, *d_salt_lens;
    BYTE *d_out_seeds;

    CudaSafeCall(hipMalloc(&d_mnemonics, count * sizeof(char*)));
    CudaSafeCall(hipMalloc(&d_salts, count * sizeof(char*)));
    CudaSafeCall(hipMalloc(&d_mnemonic_lens, count * sizeof(size_t)));
    CudaSafeCall(hipMalloc(&d_salt_lens, count * sizeof(size_t)));
    CudaSafeCall(hipMalloc(&d_out_seeds, count * SEED_SIZE));

    std::vector<char*> d_mnemonic_data(count), d_salt_data(count);
    for (int i = 0; i < count; ++i) {
        CudaSafeCall(hipMalloc(&d_mnemonic_data[i], mnemonic_lens[i]));
        CudaSafeCall(hipMemcpy(d_mnemonic_data[i], h_mnemonics[i], mnemonic_lens[i], hipMemcpyHostToDevice));
        CudaSafeCall(hipMalloc(&d_salt_data[i], salt_lens[i]));
        CudaSafeCall(hipMemcpy(d_salt_data[i], h_salts[i], salt_lens[i], hipMemcpyHostToDevice));
    }

    CudaSafeCall(hipMemcpy(d_mnemonics, d_mnemonic_data.data(), count * sizeof(char*), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_salts, d_salt_data.data(), count * sizeof(char*), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_mnemonic_lens, mnemonic_lens.data(), count * sizeof(size_t), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(d_salt_lens, salt_lens.data(), count * sizeof(size_t), hipMemcpyHostToDevice));

    int blocks = (count + threads_per_block - 1) / threads_per_block;
    pbkdf2_kernel<<<blocks, threads_per_block>>>(
        d_mnemonics, d_mnemonic_lens, d_salts, d_salt_lens,
        PBKDF2_HMAC_SHA512_ITERATIONS, d_out_seeds, count);
    CudaSafeCall(hipDeviceSynchronize());

    out_seeds.resize(count);
    for (int i = 0; i < count; ++i) {
        out_seeds[i].resize(SEED_SIZE);
        CudaSafeCall(hipMemcpy(out_seeds[i].data(), d_out_seeds + i * SEED_SIZE, SEED_SIZE, hipMemcpyDeviceToHost));
    }

    for (int i = 0; i < count; ++i) {
        hipFree(d_mnemonic_data[i]);
        hipFree(d_salt_data[i]);
    }
    hipFree(d_mnemonics);
    hipFree(d_salts);
    hipFree(d_mnemonic_lens);
    hipFree(d_salt_lens);
    hipFree(d_out_seeds);
}

std::vector<std::vector<unsigned char>> py_derive_pubkeys(
    const std::vector<std::string>& mnemonics,
    const std::string& passphrase,
    const std::vector<uint32_t>& path_indices,
    int threads_per_block = 256
) {
    initSHA512Constants();
    std::vector<ByteVec> seeds;
    gpu_pbkdf2_batch(mnemonics, passphrase, seeds, threads_per_block);

    int count = mnemonics.size();
    std::vector<std::vector<unsigned char>> privs(count), chains(count);
    std::vector<std::string> keys(count, BITCOIN_SEED);
    std::vector<ByteVec> outputs;
    hmac_sha512_batch(keys, seeds, outputs, threads_per_block);

    for (int i = 0; i < count; ++i) {
        privs[i].assign(outputs[i].begin(), outputs[i].begin() + 32);
        chains[i].assign(outputs[i].begin() + 32, outputs[i].end());
    }

    secp256k1_context* ctx = secp256k1_context_create(SECP256K1_CONTEXT_SIGN);
    for (uint32_t index : path_indices) {
        std::vector<ByteVec> input_data(count);
        std::vector<uint8_t> hardened(count);
        for (int i = 0; i < count; ++i) {
            hardened[i] = index >= 0x80000000;
            input_data[i] = hardened[i] ? privs[i] : derive_pubkey(privs[i], ctx, true);
        }

        std::vector<ByteVec> prepared(count);
        for (int i = 0; i < count; ++i) {
            prepared[i].clear();
            if (hardened[i]) prepared[i].push_back(0x00);
            prepared[i].insert(prepared[i].end(), input_data[i].begin(), input_data[i].end());
            prepared[i].push_back((index >> 24) & 0xFF);
            prepared[i].push_back((index >> 16) & 0xFF);
            prepared[i].push_back((index >> 8) & 0xFF);
            prepared[i].push_back(index & 0xFF);
        }

        std::vector<std::string> key_strs(count);
        for (int i = 0; i < count; ++i)
            key_strs[i] = std::string(reinterpret_cast<const char*>(chains[i].data()), chains[i].size());

        std::vector<ByteVec> out_hmac;
        hmac_sha512_batch(key_strs, prepared, out_hmac, threads_per_block);

        for (int i = 0; i < count; ++i) {
            std::vector<unsigned char> IL(out_hmac[i].begin(), out_hmac[i].begin() + 32);
            std::vector<unsigned char> IR(out_hmac[i].begin() + 32, out_hmac[i].end());

            if (!secp256k1_ec_seckey_verify(ctx, IL.data())) {
                privs[i].clear();
                continue;
            }

            privs[i] = add_privkeys_mod_n(privs[i], IL, ctx);
            chains[i] = IR;
        }
    }
    secp256k1_context_destroy(ctx);

    std::vector<std::vector<unsigned char>> final_pubkeys;
    secp256k1_context* ctx_final = secp256k1_context_create(SECP256K1_CONTEXT_SIGN);
    for (const auto& priv : privs) {
        if (!priv.empty())
            final_pubkeys.push_back(derive_pubkey(priv, ctx_final, false));
    }
    secp256k1_context_destroy(ctx_final);

    return final_pubkeys;
}

int main() {
    std::vector<std::string> mnemonics = {
        "aware report movie exile buyer drum poverty supreme gym oppose float elegant",
        "shock mosquito dizzy upper sniff mother promote peanut month then coin trade"
    };

    std::string passphrase = "";

    std::vector<uint32_t> path = {
        44 | 0x80000000,
        195 | 0x80000000,
        0 | 0x80000000,
        0,
        0
    };

    std::vector<std::vector<unsigned char>> pubkeys;

    try {
        pubkeys = py_derive_pubkeys(mnemonics, passphrase, path, 256); 

        for (const auto& pub : pubkeys) {
            for (unsigned char b : pub)
                std::cout << std::hex << std::setw(2) << std::setfill('0') << (int)b;
            std::cout << '\n';
        }

    } catch (const std::exception& e) {
        std::cerr << "[!] Error: " << e.what() << std::endl;
        return 1;
    }

    return 0;
}
